#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <numeric>
using namespace std;

__global__ void sum(int* input)
{
	const int tid = threadIdx.x;

	int step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads) // still alive?
		{
			const int fst = tid * step_size * 2;
			const int snd = fst + step_size;
			input[fst] += input[snd];
		}

		step_size <<= 1;
		number_of_threads >>= 1;
	}
}

int main()
{
	const int count = 8;
	const int size = count * sizeof(int);
	int h[] = {13, 27, 15, 14, 33, 4, 24, 6};

	int* d;

	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	sum <<<1, count / 2 >>>(d);

	int result;
	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);

	cout << "Sum is " << result << endl;

	//getchar();

	hipFree(d);
	delete[] h;

	return 0;
}
/*
 Sum is 136
 */
