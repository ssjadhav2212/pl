/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include<hip/hip_runtime.h>
#include <stdlib.h>
#include<time.h>

#define SIZE 10

__global__ void vectsum(int *x,int *y,int *z)
{
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	z[tid]=x[tid]+y[tid];
}

int main(void)
{
	int i;
	srand(time(NULL));
	int a[SIZE],b[SIZE],c[SIZE];

	int *dev_a,*dev_b,*dev_c;

	hipMalloc((void **)&dev_a, SIZE*sizeof(int));
	hipMalloc((void **)&dev_b, SIZE*sizeof(int));
	hipMalloc((void **)&dev_c, SIZE*sizeof(int));

	for(i=0;i<SIZE;i++)
	{
		a[i] = rand()%20+1;
	}

	printf("\nThe 1st vector is:\n");
	for(i=0;i<SIZE;i++)
	{
		printf("%d  ",a[i]);
	}

	for(i=0;i<SIZE;i++)
	{
		b[i] = rand()%20+1;
	}

	printf("\nThe 2nd vector is:\n");
	for(i=0;i<SIZE;i++)
	{
		printf("%d  ",b[i]);
	}

	hipMemcpy(dev_a,a,sizeof(a),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,sizeof(b),hipMemcpyHostToDevice);
	vectsum<<<1,SIZE>>>(dev_a,dev_b,dev_c);
	hipMemcpy(&c,dev_c,sizeof(c),hipMemcpyDeviceToHost);

	printf("\nThe result is:\n");
	for(int i=0;i<SIZE;i++)
	{
		printf("%d  ",c[i]);
	}


	return 0;
}
