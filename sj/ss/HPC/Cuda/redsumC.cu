
#include <hip/hip_runtime.h>
#include <stdio.h>

#define w 32
#define h 32
#define N w*h

__global__ void reduce(int *g_idata, int *g_odata);
void fill_array (int *a, int n);

int main( void ) {
    int a[N], b[N]; // copies of a, b, c
    int *dev_a, *dev_b; // device copies of a, b, c
    int size = N * sizeof( int ); // we need space for 512 integers

    // allocate device copies of a, b, c
    hipMalloc( (void**)&dev_a, size );
    hipMalloc( (void**)&dev_b, size );

    fill_array( a, N );
    b[0] = 0;  //initialize the first value of b to zero
    // copy inputs to device
    hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );

    dim3 blocksize(256); // create 1D threadblock
    dim3 gridsize(N/blocksize.x);  //create 1D grid

    reduce<<<gridsize, blocksize>>>(dev_a, dev_b);

    // copy device result back to host copy of c
    hipMemcpy( b, dev_b, sizeof( int ) , hipMemcpyDeviceToHost );

    printf("Reduced sum of Array elements = %d \n", b[0]);
    printf("Value should be: %d \n", ((N-1)*(N/2)));
    hipFree( dev_a );
    hipFree( dev_b );

    return 0;
}

__global__ void reduce(int *g_idata, int *g_odata) {

    __shared__ int sdata[256];

    // each thread loads one element from global to shared mem
    // note use of 1D thread indices (only) in this kernel
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    sdata[threadIdx.x] = g_idata[i];

    __syncthreads();
    // do reduction in shared mem
    for (int s=1; s < blockDim.x; s *=2)
    {
        int index = 2 * s * threadIdx.x;;

        if (index < blockDim.x)
        {
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (threadIdx.x == 0)
        atomicAdd(g_odata,sdata[0]);
}

// CPU function to generate a vector of random integers
void fill_array (int *a, int n)
{
    for (int i = 0; i < n; i++)
        a[i] = i;
}