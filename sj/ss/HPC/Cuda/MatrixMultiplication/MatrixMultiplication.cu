#include<hip/hip_runtime.h>
#include<stdio.h>

int main(void) {
    void MatrixMultiplication(float *, float *, float *, int);
    const int Width = 3;
    float M[Width*Width], N[Width*Width], P[Width*Width];
    for(int i = 0; i < (Width*Width) ; i++) {
        M[i] = rand()%10;
        N[i] = rand()%10;
        P[i] = 0;
    }
    printf("First Matrix:\n");
    for(int i = 0; i < (Width*Width) ; i=i+Width) {
    for(int j = 0; j < (Width) ; j++) {
            printf("%f \t", M[i+j]);
        }
    printf("\n");
    }
    printf("\n");
    printf("Second Matrix:\n");
    for(int i = 0; i < (Width*Width) ; i=i+Width) {
        for(int j = 0; j < (Width) ; j++) {
                printf("%f \t", N[i+j]);
            }
        printf("\n");
        }
    printf("\n");
    printf("Multiplication :\n");
    MatrixMultiplication(M, N, P, Width);
    for(int i = 0; i < (Width*Width) ; i=i+Width) {
        for(int j = 0; j < (Width) ; j++) {
                printf("%f \t", P[i+j]);
            }
        printf("\n");
        }
    int quit;
    scanf("%d",&quit);
    return 0;
}

//Matrix multiplication kernel - thread specification
__global__ void MatrixMulKernel(float *Md, float *Nd, float *Pd, int Width) {
    //2D Thread ID
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    //Pvalue stores the Pd element that is computed by the thread
    float Pvalue = 0;

    for(int k = 0; k < Width ; ++k) {
        float Mdelement = Md[ty*Width + k];
        float Ndelement = Nd[k*Width + tx];
        Pvalue += (Mdelement*Ndelement);
    }

    Pd[ty*Width + tx] = Pvalue;
}

void MatrixMultiplication(float *M, float *N, float *P, int Width) {
    int size = Width*Width*sizeof(float);
    float *Md, *Nd, *Pd;

    //Transfer M and N to device memory
    hipMalloc((void**)&Md, size);
    hipMemcpy(Md,M,size,hipMemcpyHostToDevice);
    hipMalloc((void**)&Nd, size);
    hipMemcpy(Nd,N,size,hipMemcpyHostToDevice);

    //Allocate P on the device
    hipMalloc((void**)&Pd,size);

    //Setup the execution configuration
    dim3 dimBlock(Width,Width);
    dim3 dimGrid(1,1);

    //Launch the device computation threads!
    MatrixMulKernel<<<dimGrid,dimBlock>>>(Md,Nd,Pd,Width);

    //Transfer P from device to host
    hipMemcpy(P,Pd,size,hipMemcpyDeviceToHost);

    //Free device matrices
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);
}
