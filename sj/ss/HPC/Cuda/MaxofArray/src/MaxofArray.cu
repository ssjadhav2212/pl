/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include<hip/hip_runtime.h>
#include <stdlib.h>
#include<time.h>

#define SIZE 6

__global__ void max(int *a, int *c)
{
	extern __shared__ int sdata[];
	unsigned int tid=threadIdx.x;
	unsigned int i=blockIdx.x*blockDim.x+threadIdx.x;

	sdata[tid]=a[i];

	__syncthreads();

	for(unsigned int s=blockDim.x/2; s>=1; s=s/2)
	{
		if(tid<s)
		{
			if(sdata[tid]<sdata[tid+s])
			{
				sdata[tid]=sdata[tid+s];
			}
		}
		__syncthreads();
	}
	if(tid==0)
	{
		*c=sdata[0];
	}
}

__global__ void min(int *a, int *c)
{
	extern __shared__ int sdata[];
	unsigned int tid=threadIdx.x;
	unsigned int i=blockIdx.x*blockDim.x+threadIdx.x;

	sdata[tid]=a[i];

	__syncthreads();

	for(unsigned int s=blockDim.x/2; s>=1; s=s/2)
	{
		if(tid<s)
		{
			if(sdata[tid]>sdata[tid+s])
			{
				sdata[tid]=sdata[tid+s];
			}
		}
		__syncthreads();
	}
	if(tid==0)
	{
		*c=sdata[0];
	}
}

__global__ void calcavg(int *a, float *avg)
{
	*avg=0;
	for(int i=0;i<SIZE;i++)
	{
		*avg=*avg+a[i];
	}
	*avg=*avg/SIZE;
}


int main(void)
{
	int i;
	srand(time(NULL));
	int a[SIZE];
	int c,d;
	float avg;

	int *dev_a,*dev_c,*dev_d;
	float *average;

	hipMalloc((void **)&dev_a, SIZE*sizeof(int));
	hipMalloc(&dev_c, sizeof(int));
	hipMalloc(&dev_d, sizeof(int));
	hipMalloc(&average, sizeof(float));

	for(i=0;i<SIZE;i++)
	{
		a[i] = rand()%20+1;
	}

	printf("\nThe array is:\n");
	for(i=0;i<SIZE;i++)
	{
		printf("%d  ",a[i]);
	}


	hipMemcpy(dev_a,a,sizeof(a),hipMemcpyHostToDevice);
	max<<<1,SIZE>>>(dev_a,dev_c);
	min<<<1,SIZE>>>(dev_a,dev_d);
	calcavg<<<1,1>>>(dev_a,average);
	hipMemcpy(&c,dev_c,sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(&d,dev_d,sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(&avg,average,sizeof(float),hipMemcpyDeviceToHost);

	printf("\nMax is: %d",c);
	printf("\nMin is: %d",d);
	printf("\nAverage is: %f",avg);
	hipFree(dev_a);
	hipFree(dev_c);
	hipFree(dev_d);
	hipFree(average);
	return 0;
}
