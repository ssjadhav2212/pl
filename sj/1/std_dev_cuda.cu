
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#include<iostream>

#define w 256
#define h 256

#define N w*h

using namespace std;

__global__ void reduce(int*,int*,int*);

int main(void)
{
	int* hostA = (int*)malloc(N*sizeof(int));
	int* hostB = (int*)malloc(N*sizeof(int));

	int* hostMean = (int*)malloc(sizeof(int));

	*hostMean = 32767;

	int* deviceA; int *deviceB;int*deviceMean;

	hipMalloc(&deviceA,sizeof(int)*N);
	hipMalloc(&deviceB,sizeof(int)*N);
	hipMalloc(&deviceMean,sizeof(int));


	//randomly generate array hostA
	srand(time(0));
	int i;

	//initialize host vector by random elements
	for(i=0;i<N;i++)
	{
		hostA[i] = i;
	}

	hostB[0]=0.0;

	hipMemcpy(deviceA,hostA,N*sizeof(int),hipMemcpyHostToDevice);

	hipMemcpy(deviceB,hostB,N*sizeof(int),hipMemcpyHostToDevice);

	hipMemcpy(deviceMean,hostMean,sizeof(int),hipMemcpyHostToDevice);

	dim3 blocksize(256);
	dim3 gridsize(N/blocksize.x);

	float gpu_elapsed_time;
	hipEvent_t gpu_start,gpu_stop;

	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);

	hipEventRecord(gpu_start,0);


	reduce<<<gridsize,blocksize>>>(deviceA,deviceB,deviceMean);

	hipDeviceSynchronize();

	hipMemcpy(hostB,deviceB,sizeof(int),hipMemcpyDeviceToHost);

	hipEventRecord(gpu_stop, 0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);

    
    
	

	double std_dev = pow(hostB[0]/(N),0.5);
	cout<<"Reduced array standard deviation   is = "<<std_dev<<endl;

	std::cout<<"The gpu took: "<<gpu_elapsed_time<<" milli-seconds"<<std::endl;

	



     clock_t cpu_start = clock();

	int sum=0;
	for(int i=0;i<N;i++){
	sum = sum  + int(pow((hostA[i] - (*hostMean)),2.0));
	}

	//cout<<"sum == "<<sum<<endl;

	double std_dev_actual = pow(sum/(N),0.5);

	printf("Actual value of standard deviation should be: %f \n", std_dev_actual);

	clock_t cpu_stop = clock();
	clock_t cpu_elapsed_time = 1000*(cpu_stop - cpu_start)/CLOCKS_PER_SEC;

	

	std::cout<<"The cpu took: "<<cpu_elapsed_time<<" milli-seconds"<<std::endl;



	hipFree(deviceA);
	hipFree(deviceB);


	delete[] hostB;
	delete[] hostA;









}


__global__ void reduce(int* input,int* output,int* mean)
{
	__shared__ int shared_data[256];

	int  i = blockIdx.x*blockDim.x+threadIdx.x;

	shared_data[threadIdx.x] = int( pow(double(input[i]- *mean),2.0));

	__syncthreads();

	for(int s=1;s<blockDim.x;s*=2)
	{
		int index = 2 * s * threadIdx.x;;

        if (index < blockDim.x)
        {
            shared_data[index] += shared_data[index + s];
        }
        __syncthreads();
	}

	if (threadIdx.x == 0)
        atomicAdd(output,shared_data[0]);
}