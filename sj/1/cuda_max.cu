
#include <hip/hip_runtime.h>
#include<math.h>
#include<time.h>
#include<stdexcept>
#include<iostream>
#include<cstdlib> //for abs(x)
#include<stdio.h>


using namespace std;

__global__ void findMax(int* A,int* current_max,int* mutex,unsigned int n);



int main()
{
	const int NUMBER_OF_ELEMENTS = 1024*1024*20;

	int* hostA = (int*)malloc(NUMBER_OF_ELEMENTS*sizeof(int));

	int* hostMax = (int*)malloc(sizeof(int));

	*hostMax = -1;

	srand(time(0));
	int i,j;

	//initialize host vector by random elements
	for(i=0;i<NUMBER_OF_ELEMENTS;i++)
	{
		hostA[i] = NUMBER_OF_ELEMENTS*rand() / RAND_MAX/123;
		
	}
	int* deviceA,*deviceMax,*deviceMutex;

	hipMalloc(&deviceA,NUMBER_OF_ELEMENTS*sizeof(int));
	hipMalloc(&deviceMax,sizeof(int));
	hipMalloc(&deviceMutex,sizeof(int));

	hipMemset(deviceMax,-1,sizeof(int));
	hipMemset(deviceMutex,0,sizeof(int));

	hipMemcpy(deviceA,hostA,NUMBER_OF_ELEMENTS*sizeof(int),hipMemcpyHostToDevice);

	//set up timing variables

	float gpu_elapsed_time;
	hipEvent_t gpu_start,gpu_stop;

	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);

	hipEventRecord(gpu_start,0);


	findMax<<<256,256>>>(deviceA,deviceMax,deviceMutex,NUMBER_OF_ELEMENTS);

	hipDeviceSynchronize();

	hipMemcpy(hostMax,deviceMax,sizeof(int),hipMemcpyDeviceToHost);
	hipEventRecord(gpu_stop, 0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);

	cout<<"Answer by CUDA for MAX is = "<<*hostMax<<endl;
	std::cout<<"The gpu took: "<<gpu_elapsed_time<<" milli-seconds"<<std::endl;

	






	clock_t cpu_start = clock();

	int maxx = -1;

	for(int i=0;i<NUMBER_OF_ELEMENTS;i++)
	{
	if(hostA[i]>maxx)
	    maxx = hostA[i];
	}

	clock_t cpu_stop = clock();
	clock_t cpu_elapsed_time = 1000*(cpu_stop - cpu_start)/CLOCKS_PER_SEC;

	cout<<"Expected max value is = "<<maxx<<endl;

	std::cout<<"The cpu took: "<<cpu_elapsed_time<<" milli-seconds"<<std::endl;


	hipFree(deviceA);

	delete[] hostA;

	return hipDeviceSynchronize();
    	




    

    	



}
__global__ void findMax(int* A,int* current_max,int* mutex,unsigned int n)
{
		//printf("threadIdx.x = %d and blockIdx = %d and gridDim.x = %d\n",threadIdx.x,blockIdx.x,gridDim.x);

		unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
		unsigned int stride = gridDim.x*blockDim.x;

		unsigned int offset = 0;

		__shared__ int cache[256];

		int temp = -1;
		while(index+offset<n)
		{
		temp = fmaxf(temp,A[index+offset]);
		offset+=stride;
		}

		cache[threadIdx.x]=temp;

		__syncthreads();


		//reduction
		//printf("blockDim.x = %d\n",blockDim.x/2);
		unsigned int i=blockDim.x/2;
		while(i!=0)
		{
		if(threadIdx.x<i)
		{
		cache[threadIdx.x] = fmaxf(cache[threadIdx.x],cache[threadIdx.x+i]);

		}
		__syncthreads();
		i/=2;
		}

		if(threadIdx.x ==0)
		while(atomicCAS(mutex,0,1)!=0);
		*current_max = fmaxf(*current_max,cache[0]);
		atomicExch(mutex,0);


}


