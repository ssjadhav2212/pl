
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#include<iostream>
#define w 256
#define h 256

#define N w*h

using namespace std;

__global__ void reduce(int*,int*);

int main(void)
{
	int* hostA = (int*)malloc(N*sizeof(int));
	int* hostB = (int*)malloc(N*sizeof(int));

	int* deviceA,*deviceB;

	hipMalloc(&deviceA,sizeof(int)*N);
	hipMalloc(&deviceB,sizeof(int)*N);


	//randomly generate array hostA
	srand(time(0));
	int i;

	//initialize host vector by random elements
	for(i=0;i<N;i++)
	{
		hostA[i] = i;
	}

	hostB[0]=0;

	hipMemcpy(deviceA,hostA,N*sizeof(int),hipMemcpyHostToDevice);

	hipMemcpy(deviceB,hostB,N*sizeof(int),hipMemcpyHostToDevice);

	dim3 blocksize(256);
	dim3 gridsize(N/blocksize.x);

	reduce<<<gridsize,blocksize>>>(deviceA,deviceB);

	hipDeviceSynchronize();

	hipMemcpy(hostB,deviceB,sizeof(int),hipMemcpyDeviceToHost);

	int mean = hostB[0]/(N);
	cout<<"Reduced array mean  is = "<<mean<<endl;

	printf("Actual value of mean should be: %d \n", (N-1)/2);


	hipFree(deviceA);
	hipFree(deviceB);


	delete[] hostB;
	delete[] hostA;









}


__global__ void reduce(int* input,int* output)
{
	__shared__ int shared_data[256];

	int  i = blockIdx.x*blockDim.x+threadIdx.x;

	shared_data[threadIdx.x]=input[i];

	__syncthreads();

	for(int s=1;s<blockDim.x;s*=2)
	{
		int index = 2 * s * threadIdx.x;;

        if (index < blockDim.x)
        {
            shared_data[index] += shared_data[index + s];
        }
        __syncthreads();
	}

	if (threadIdx.x == 0)
        atomicAdd(output,shared_data[0]);
}