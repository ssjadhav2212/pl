
#include <hip/hip_runtime.h>
#include<math.h>
#include<time.h>
#include<stdexcept>
#include<iostream>
#include<cstdlib> //for abs(x)
#include<stdio.h>
#include<math.h>


using namespace std;

__global__ void findMin(int* A,int* current_min,int* mutex,unsigned int n);



int main()
{
	const int NUMBER_OF_ELEMENTS = 1024*1024*20;

	int* hostA = (int*)malloc(NUMBER_OF_ELEMENTS*sizeof(int));

	int* hostMin = (int*)malloc(sizeof(int));

	*hostMin = 1230000;

	srand(time(0));
	int i;

	//initialize host vector by random elements
	for(i=0;i<NUMBER_OF_ELEMENTS;i++)
	{
		int temp = rand() % 1230000;
		if(temp<0){
		temp*=-1;
		temp = temp% 1230000;
		}
		else if(temp==0)
				temp=34;
		hostA[i] = temp;
		
	}
	int* deviceA,*deviceMin,*deviceMutex;

	hipMalloc(&deviceA,NUMBER_OF_ELEMENTS*sizeof(int));
	hipMalloc(&deviceMin,sizeof(int));
	hipMalloc(&deviceMutex,sizeof(int));

	hipMemcpy(deviceMin,hostMin,sizeof(int),hipMemcpyHostToDevice);
	hipMemset(deviceMutex,0,sizeof(int));

	hipMemcpy(deviceA,hostA,NUMBER_OF_ELEMENTS*sizeof(int),hipMemcpyHostToDevice);

	//set up timing variables

	float gpu_elapsed_time;
	hipEvent_t gpu_start,gpu_stop;

	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);

	hipEventRecord(gpu_start,0);


	findMin<<<256,256>>>(deviceA,deviceMin,deviceMutex,NUMBER_OF_ELEMENTS);

	hipDeviceSynchronize();

	hipMemcpy(hostMin,deviceMin,sizeof(int),hipMemcpyDeviceToHost);
	hipEventRecord(gpu_stop, 0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);

	cout<<"Answer by CUDA for MIN is = "<<*hostMin<<endl;
	std::cout<<"The gpu took: "<<gpu_elapsed_time<<" milli-seconds"<<std::endl;

	






	clock_t cpu_start = clock();

	int minn = 1230000;

	for(int i=0;i<NUMBER_OF_ELEMENTS;i++)
	{
	if(hostA[i]<minn)
	    minn = hostA[i];
	}

	clock_t cpu_stop = clock();
	clock_t cpu_elapsed_time = 1000*(cpu_stop - cpu_start)/CLOCKS_PER_SEC;

	cout<<"Expected min value is = "<<minn<<endl;

	std::cout<<"The cpu took: "<<cpu_elapsed_time<<" milli-seconds"<<std::endl;


	hipFree(deviceA);

	delete[] hostA;

	return hipDeviceSynchronize();
    	




    

    	



}
__global__ void findMin(int* A,int* current_min,int* mutex,unsigned int n)
{
		//printf("threadIdx.x = %d and blockIdx = %d and gridDim.x = %d\n",threadIdx.x,blockIdx.x,gridDim.x);

		unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
		unsigned int stride = gridDim.x*blockDim.x;

		unsigned int offset = 0;

		__shared__ int cache[256];

		int temp = 1230000;
		while(index+offset<n)
		{
		//printf("A[i] = %d and current temp = %d\n",A[index+offset],temp);
		temp = fminf(temp,A[index+offset]);
		//printf("temp == %d\n",temp);
		offset+=stride;
		}

		cache[threadIdx.x]=temp;

		__syncthreads();


		//reduction
		//printf("blockDim.x = %d\n",blockDim.x/2);
		unsigned int i=blockDim.x/2;
		while(i!=0)
		{
		if(threadIdx.x<i)
		{
		cache[threadIdx.x] = fminf(cache[threadIdx.x],cache[threadIdx.x+i]);

		}
		__syncthreads();
		i/=2;
		}

		if(threadIdx.x == 0)
		while(atomicCAS(mutex,0,1)!=0);
		//printf("current_min before = %d\n",*current_min);
		*current_min = fminf(*current_min,cache[0]);
		//printf("current_min = %d\n",*current_min);
		atomicExch(mutex,0);


}


