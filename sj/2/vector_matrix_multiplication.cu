
#include <hip/hip_runtime.h>
#include<math.h>
#include<time.h>
#include<stdexcept>
#include<iostream>
#include<cstdlib> //for abs(x)
#include<stdio.h>

using namespace std;

int NUMBER_OF_ELEMENTS  = 1<<12;
int SIZE  = NUMBER_OF_ELEMENTS*sizeof(int);
int VECTOR_SIZE = 1<<4;

__global__ void kernel_multiplication( int* A,  int* B, int* C,int N,int M);


int main()
{

   //allocate memory for host vectors

	int* hostA = (int*)malloc(VECTOR_SIZE*sizeof(int));
	int* hostB = (int*)malloc(SIZE*VECTOR_SIZE);
	int* hostC = (int*)malloc(VECTOR_SIZE*sizeof(int));

	int* deviceA,*deviceB,*deviceC;

	srand(time(0));
	int i,j;

	//initialize host vector by random elements
	for(i=0;i<VECTOR_SIZE;i++)
	{
		hostA[i] = rand();
		
	}

	//initialize matrix by random elements
	for(i=0;i<NUMBER_OF_ELEMENTS;i++)
	{
		for(j=0;j<VECTOR_SIZE;j++)
		{
			hostB[i*VECTOR_SIZE+j] = rand();
		}
	}




	//allocate memory for device vectors

	hipMalloc(&deviceA,VECTOR_SIZE*sizeof(int));
	hipMalloc(&deviceB,NUMBER_OF_ELEMENTS*VECTOR_SIZE*sizeof(int));
	hipMalloc(&deviceC,VECTOR_SIZE*sizeof(int));

	//kernel function
	
	hipMemcpy(deviceA,hostA,VECTOR_SIZE*sizeof(int),hipMemcpyHostToDevice);
	
	hipMemcpy(deviceB,hostB,SIZE*VECTOR_SIZE,hipMemcpyHostToDevice);

	kernel_multiplication<<<NUMBER_OF_ELEMENTS,1>>>(deviceA,deviceB,deviceC,NUMBER_OF_ELEMENTS,VECTOR_SIZE);


	hipDeviceSynchronize();

    hipMemcpy(hostC,deviceC,VECTOR_SIZE*sizeof(int),hipMemcpyDeviceToHost);


	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);

	double error = 0;

    int* answer = (int*) malloc(VECTOR_SIZE*sizeof(int));
	for(int i=0;i<NUMBER_OF_ELEMENTS;i++)
	{
	int sum = 0;
	for(int j=0;j<VECTOR_SIZE;j++)
	{
		sum += hostA[j]*hostB[i*VECTOR_SIZE+j];
	}
	answer[i] = sum;
	}

	for(int k=0;k<VECTOR_SIZE;k++)
	{
	//cout<<k<<")"<< "Expected value = "<<answer[k]<<" Actual value = "<<hostC[k]<<"\n";
	error += double(abs(answer[k]-hostC[k]));
	}

	error=sqrt(error);
	cout<<"error = "<<error<<"\n";

	delete[] hostA;
    delete[] hostB;
    delete[] hostC;



    return hipDeviceSynchronize();

}




__global__ void kernel_multiplication( int* A,  int* B, int* C, int N,int M)
{
	int index =  threadIdx.x + blockIdx.x * blockDim.x;
	int sum = 0;

	//printf("index = %d  and blockId is %d\n",index,blockIdx.x);


	if(index<N)
	{
		for(int i=0;i<M;i++)
		sum+=A[i]*B[(index*M)+i];
		C[index] = sum;

		

		//printf("index = %d and value is %d\n",index,C[index]);
	}
}